#include "hip/hip_runtime.h"
/**
* Copyright (c) 2020 Stanford University and the Authors
* Authors: Peter Eastman
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
* 
* The above copyright notice and this permission notice shall be included in all
* copies or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/

#include "CudaCFConv.h"
#include <cstring>
#include <stdexcept>

using namespace std;

#define CHECK_RESULT(result) \
if (result != hipSuccess) { \
    throw runtime_error(string("Encountered error ")+hipGetErrorName(result)+" at "+__FILE__+":"+to_string(__LINE__));\
}

const float Pi = (float) M_PI;

CudaCFConvNeighbors::CudaCFConvNeighbors(int numAtoms, float cutoff, bool periodic) : CFConvNeighbors(numAtoms, cutoff, periodic),
                                         positions(0), periodicBoxVectors(0), neighbors(0), neighborCount(0), neighborDistances(0) {
    CHECK_RESULT(hipMallocManaged(&positions, numAtoms*sizeof(float3)));
    CHECK_RESULT(hipMallocManaged(&periodicBoxVectors, 9*sizeof(float)));
    CHECK_RESULT(hipMallocManaged(&neighbors, numAtoms*numAtoms*sizeof(int2)));
    CHECK_RESULT(hipMallocManaged(&neighborCount, sizeof(int)));
    CHECK_RESULT(hipMallocManaged(&neighborDistances, numAtoms*numAtoms*sizeof(float)));
    CHECK_RESULT(hipMallocManaged(&neighborDeltas, numAtoms*numAtoms*sizeof(float3)));

    // We will limit how many thread blocks we try to launch based on the size of the GPU.

    int device;
    CHECK_RESULT(hipGetDevice(&device));
    CHECK_RESULT(hipDeviceGetAttribute(&numMultiprocessors, hipDeviceAttributeMultiprocessorCount, device));
}

CudaCFConvNeighbors::~CudaCFConvNeighbors() {
    if (positions != 0)
        hipFree(positions);
    if (periodicBoxVectors != 0)
        hipFree(periodicBoxVectors);
    if (neighbors != 0)
        hipFree(neighbors);
    if (neighborCount != 0)
        hipFree(neighborCount);
    if (neighborDistances != 0)
        hipFree(neighborDistances);
    if (neighborDeltas != 0)
        hipFree(neighborDeltas);
}

template <bool PERIODIC, bool TRICLINIC>
__device__ void computeDisplacement(float3 pos1, float3 pos2, float3& delta, float& r2, const float* periodicBoxVectors, float3 invBoxSize) {
    delta.x = pos2.x-pos1.x;
    delta.y = pos2.y-pos1.y;
    delta.z = pos2.z-pos1.z;
    if (PERIODIC) {
        if (TRICLINIC) {
            float scale3 = roundf(delta.z*invBoxSize.z);
            delta.x -= scale3*periodicBoxVectors[2*3+0];
            delta.y -= scale3*periodicBoxVectors[2*3+1];
            delta.z -= scale3*periodicBoxVectors[2*3+2];
            float scale2 = roundf(delta.y*invBoxSize.y);
            delta.x -= scale2*periodicBoxVectors[1*3+0];
            delta.y -= scale2*periodicBoxVectors[1*3+1];
            float scale1 = roundf(delta.x*invBoxSize.x);
            delta.x -= scale1*periodicBoxVectors[0*3+0];
        }
        else {
            delta.x -= roundf(delta.x*invBoxSize.x)*periodicBoxVectors[0*3+0];
            delta.y -= roundf(delta.y*invBoxSize.y)*periodicBoxVectors[1*3+1];
            delta.z -= roundf(delta.z*invBoxSize.z)*periodicBoxVectors[2*3+2];
        }
    }
    r2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
}

template <bool PERIODIC, bool TRICLINIC>
__global__ void buildNeighborList(int numAtoms, float cutoff, int2* neighbors, int* neighborCount, float* neighborDistances,
            float3* neighborDeltas, const float3* positions, const float* periodicBoxVectors) {
    const int warp = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int indexInWarp = threadIdx.x%32;
    const int numWarps = (gridDim.x*blockDim.x)/32;
    const int warpMask = (1<<indexInWarp)-1;
    const float3 invBoxSize = (PERIODIC ? make_float3(1/periodicBoxVectors[0], 1/periodicBoxVectors[4], 1/periodicBoxVectors[8]) : make_float3(0, 0, 0));
    const float cutoff2 = cutoff*cutoff;

    // Each warp loops over atoms.

    for (int atom1 = warp; atom1 < numAtoms; atom1 += numWarps) {
        float3 pos1 = positions[atom1];

        // The threads in the warp loop over second atoms.

        for (int atom2 = atom1+indexInWarp; atom2 < numAtoms; atom2 += 32) {
            float3 pos2 = positions[atom2];
            float3 delta;
            float r2;
            computeDisplacement<PERIODIC, TRICLINIC>(pos1, pos2, delta, r2, periodicBoxVectors, invBoxSize);
            bool isNeighbor = (r2 < cutoff2 && atom1 != atom2);
            int neighborFlags = __ballot_sync(0xFFFFFFFF, isNeighbor);
            int startIndex = 0;
            if (indexInWarp == 0)
                startIndex = atomicAdd(neighborCount, __popc(neighborFlags));
            startIndex = __shfl_sync(0xFFFFFFFF, startIndex, 0);
            if (isNeighbor) {
                int index = startIndex + __popc(neighborFlags&warpMask);
                neighbors[index] = make_int2(atom1, atom2);
                neighborDistances[index] = sqrtf(r2);
                neighborDeltas[index] = delta;
            }
        }
    }
}

void CudaCFConvNeighbors::build(const float* positions, const float* periodicBoxVectors) {
    // If necessary, copy the positions to the device.

    hipPointerAttribute_t attrib;
    hipError_t result = hipPointerGetAttributes(&attrib, positions);
    if (result != hipSuccess || attrib.devicePointer == 0) {
        CHECK_RESULT(hipMemcpyAsync(this->positions, positions, 3*getNumAtoms()*sizeof(float), hipMemcpyDefault));
        devicePositions = this->positions;
    }
    else
        devicePositions = (float*) attrib.devicePointer;

    // We'll need to access the box vectors on both host and device.  Figure out the most
    // efficient way of doing that.

   const float* hostBoxVectors;
    if (getPeriodic()) {
        result = hipPointerGetAttributes(&attrib, periodicBoxVectors);
        if (result != hipSuccess || attrib.devicePointer == 0) {
            CHECK_RESULT(hipMemcpyAsync(this->periodicBoxVectors, periodicBoxVectors, 9*sizeof(float), hipMemcpyDefault));
            hostBoxVectors = periodicBoxVectors;
            deviceBoxVectors = this->periodicBoxVectors;
        }
        else {
            if (attrib.hostPointer == 0) {
                CHECK_RESULT(hipMemcpy(this->periodicBoxVectors, periodicBoxVectors, 9*sizeof(float), hipMemcpyDefault));
                hostBoxVectors = this->periodicBoxVectors;
            }
            else
                hostBoxVectors = periodicBoxVectors;
            deviceBoxVectors = (float*) attrib.devicePointer;
        }
    }

    // Determine whether we have a rectangular or triclinic periodic box.
    
    triclinic = false;
    if (getPeriodic())
        for (int i = 0 ; i < 3; i++)
            for (int j = 0; j < 3; j++)
                if (i != j && hostBoxVectors[3*i+j] != 0)
                    triclinic = true;

    // Build the neighbor list.

    CHECK_RESULT(hipMemsetAsync(neighborCount, 0, sizeof(int)));
    int blockSize = 192;
    int numBlocks = min(numMultiprocessors*2, getNumAtoms());
    if (getPeriodic()) {
        if (triclinic)
            buildNeighborList<true, true><<<numBlocks, blockSize>>>(getNumAtoms(), getCutoff(), neighbors, neighborCount, neighborDistances, neighborDeltas, (float3*) devicePositions, deviceBoxVectors);
        else
            buildNeighborList<true, false><<<numBlocks, blockSize>>>(getNumAtoms(), getCutoff(), neighbors, neighborCount, neighborDistances, neighborDeltas, (float3*) devicePositions, deviceBoxVectors);
    }
    else
        buildNeighborList<false, false><<<numBlocks, blockSize>>>(getNumAtoms(), getCutoff(), neighbors, neighborCount, neighborDistances, neighborDeltas, (float3*) devicePositions, deviceBoxVectors);
}

CudaCFConv::CudaCFConv(int numAtoms, int width, int numGaussians, float cutoff, bool periodic, float gaussianWidth,
                       ActivationFunction activation, const float* w1, const float* b1, const float* w2, const float* b2) :
            CFConv(numAtoms, width, numGaussians, cutoff, periodic, gaussianWidth, activation),
            input(0), output(0), inputDeriv(0), positionDeriv(0), w1(0), b1(0), w2(0), b2(0) {
    // Allocate memory on the device for the layer parameters.

    CHECK_RESULT(hipMallocManaged(&this->w1, numGaussians*width*sizeof(float)));
    CHECK_RESULT(hipMallocManaged(&this->w2, width*width*sizeof(float)));
    CHECK_RESULT(hipMallocManaged(&this->b1, width*sizeof(float)));
    CHECK_RESULT(hipMallocManaged(&this->b2, width*sizeof(float)));

    // Copy the layer parameters to device memory.  The weight matrices are stored in transposed order,
    // since that allows more efficient access in the kernels.

    CHECK_RESULT(hipMemcpyAsync(this->b1, b1, width*sizeof(float), hipMemcpyDefault));
    CHECK_RESULT(hipMemcpyAsync(this->b2, b2, width*sizeof(float), hipMemcpyDefault));
    for (int i = 0; i < numGaussians; i++)
        for (int j = 0; j < width; j++)
            this->w1[i*width+j] = w1[i+j*numGaussians];
    for (int i = 0; i < width; i++)
        for (int j = 0; j < width; j++)
            this->w2[i*width+j] = w2[i+j*width];

    // We will limit how many thread blocks we try to launch based on the size of the GPU.

    int device;
    CHECK_RESULT(hipGetDevice(&device));
    CHECK_RESULT(hipDeviceGetAttribute(&numMultiprocessors, hipDeviceAttributeMultiprocessorCount, device));
}

CudaCFConv::~CudaCFConv() {
    if (input != 0)
        hipFree(input);
    if (output != 0)
        hipFree(output);
    if (inputDeriv != 0)
        hipFree(inputDeriv);
    if (positionDeriv != 0)
        hipFree(positionDeriv);
    if (w1 != 0)
        hipFree(w1);
    if (b1 != 0)
        hipFree(b1);
    if (w2 != 0)
        hipFree(w2);
    if (b2 != 0)
        hipFree(b2);
}

float* CudaCFConv::ensureOnDevice(float* arg, float*& deviceMemory, int size) {
    hipPointerAttribute_t attrib;
    hipError_t result = hipPointerGetAttributes(&attrib, arg);
    if (result != hipSuccess || attrib.devicePointer == 0) {
        if (deviceMemory == 0)
            CHECK_RESULT(hipMallocManaged(&deviceMemory, size));
        CHECK_RESULT(hipMemcpyAsync(deviceMemory, arg, size, hipMemcpyDefault));
        return deviceMemory;
    }
    return (float*) attrib.devicePointer;
}

const float* CudaCFConv::ensureOnDevice(const float* arg, float*& deviceMemory, int size) {
    hipPointerAttribute_t attrib;
    hipError_t result = hipPointerGetAttributes(&attrib, arg);
    if (result != hipSuccess || attrib.devicePointer == 0) {
        if (deviceMemory == 0)
            CHECK_RESULT(hipMallocManaged(&deviceMemory, size));
        CHECK_RESULT(hipMemcpyAsync(deviceMemory, arg, size, hipMemcpyDefault));
        return deviceMemory;
    }
    return (const float*) attrib.devicePointer;
}

static __device__ float cutoffFunction(float r, float rc) {
    return 0.5f * cosf(Pi*r/rc) + 0.5f;
}

static __device__ float cutoffDeriv(float r, float rc) {
    return -(0.5f*Pi/rc) * sinf(Pi*r/rc);
}

static __device__ int warpReduceMin(int x) {
    for (int offset = 16; offset > 0; offset /= 2)
        x = min(x, __shfl_down_sync(0xFFFFFFFF, x, offset));
    return __shfl_sync(0xFFFFFFFF, x, 0);
}

static __device__ int warpReduceMax(int x) {
    for (int offset = 16; offset > 0; offset /= 2)
        x = max(x, __shfl_down_sync(0xFFFFFFFF, x, offset));
    return __shfl_sync(0xFFFFFFFF, x, 0);
}

__global__ void computeCFConv(int numAtoms, int numGaussians, int width, float cutoff, float gaussianWidth, int activation,
            const int2* neighbors, const int* neighborCount, const float* neighborDistance, const float* input,
            float* output, const float* w1, const float* b1, const float* w2, const float* b2) {
    const int warp = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int indexInWarp = threadIdx.x%32;
    const int numWarps = (gridDim.x*blockDim.x)/32;
    const int warpInBlock = threadIdx.x/32;
    const int numWarpsInBlock = blockDim.x/32;
    const int tempSize = max(numGaussians, width);
    extern __shared__ float tempArrays[];
    float* temp1 = &tempArrays[tempSize*warpInBlock];
    float* temp2 = &tempArrays[tempSize*(numWarpsInBlock+warpInBlock)];

    // Each warp loops over pairs of atoms.

    for (int pair = warp; pair < *neighborCount; pair += numWarps) {
        int atom1 = neighbors[pair].x;
        int atom2 = neighbors[pair].y;
        float r = neighborDistance[pair];

        // Compute the Gaussian basis functions and store them in temp1.

        int first = numGaussians, last = 0;
        for (int i = indexInWarp; i < numGaussians; i += 32) {
            float gaussianPos = i*cutoff/(numGaussians-1);
            float x = (r-gaussianPos)/gaussianWidth;
            temp1[i] = expf(-0.5f*x*x);
            if (temp1[i] > 1e-15) {
                if (first == numGaussians)
                    first = i;
                last = i;
            }
        }
        first = warpReduceMin(first);
        last = warpReduceMax(last);

        // Apply the first dense layer, storing the result in temp2.

        for (int i = indexInWarp; i < width; i += 32) {
            float sum = b1[i];
            for (int j = first; j <= last; j++)
                sum += temp1[j]*w1[i+j*width];
            if (activation == 0)
                temp2[i] = logf(0.5f*expf(sum) + 0.5f);
            else
                temp2[i] = tanhf(sum);
        }
        __syncwarp();

        // Apply the second dense layer, storing the result in temp1.

        float cutoffScale = cutoffFunction(r, cutoff);
        for (int i = indexInWarp; i < width; i += 32) {
            float sum = b2[i];
            for (int j = 0; j < width; j++)
                sum += temp2[j]*w2[i+j*width];
            temp1[i] = cutoffScale*sum;
        }
        __syncwarp();

        // Add it to the output.

        for (int i = indexInWarp; i < width; i += 32) {
            atomicAdd(&output[atom1*width+i], temp1[i]*input[atom2*width+i]);
            atomicAdd(&output[atom2*width+i], temp1[i]*input[atom1*width+i]);
        }
    }
}

void CudaCFConv::compute(const CFConvNeighbors& neighbors, const float* positions, const float* periodicBoxVectors,
                const float* input, float* output) {
    // Get device pointers to all the data we need, copying it if necessary.

    const float* deviceInput = ensureOnDevice(input, this->input, getNumAtoms()*getWidth()*sizeof(float));
    float* deviceOutput = ensureOnDevice(output, this->output, getNumAtoms()*getWidth()*sizeof(float));

    // Clear the output array.

    CHECK_RESULT(hipMemsetAsync(deviceOutput, 0, getNumAtoms()*getWidth()*sizeof(float)));

    // Invoke the kernel.

    const int blockSize = 512;
    const CudaCFConvNeighbors& cudaNeighbors = dynamic_cast<const CudaCFConvNeighbors&>(neighbors);
    const int numBlocks = numMultiprocessors*2;
    const int tempSize = max(getNumGaussians(), getWidth())*(blockSize/32);
    computeCFConv<<<numBlocks, blockSize, 2*tempSize*sizeof(float)>>>(getNumAtoms(), getNumGaussians(),
        getWidth(), getCutoff(), getGaussianWidth(), getActivation(), cudaNeighbors.getNeighbors(),
        cudaNeighbors.getNeighborCount(), cudaNeighbors.getNeighborDistances(), deviceInput,
        deviceOutput, w1, b1, w2, b2);

    // If necessary, copy the output.

    if (deviceOutput == this->output)
        CHECK_RESULT(hipMemcpy(output, deviceOutput, getNumAtoms()*getWidth()*sizeof(float), hipMemcpyDefault));
}

template <bool PERIODIC, bool TRICLINIC>
__global__ void backpropCFConv(int numAtoms, int numGaussians, int width, float cutoff, float gaussianWidth,
            int activation, const int2* neighbors, const int* neighborCount, const float3* neighborDeltas,
            const float* input, const float* outputDeriv, float* inputDeriv, float* positionDeriv, const float* w1,
            const float* b1, const float* w2, const float* b2) {
    const int warp = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int indexInWarp = threadIdx.x%32;
    const int numWarps = (gridDim.x*blockDim.x)/32;
    const int warpInBlock = threadIdx.x/32;
    const int numWarpsInBlock = blockDim.x/32;
    const int tempSize = max(numGaussians, width);
    extern __shared__ float tempArrays[];
    float* temp1 = &tempArrays[tempSize*warpInBlock];
    float* temp2 = &tempArrays[tempSize*(numWarpsInBlock+warpInBlock)];
    float* dtemp1 = &tempArrays[tempSize*(2*numWarpsInBlock+warpInBlock)];
    float* dtemp2 = &tempArrays[tempSize*(3*numWarpsInBlock+warpInBlock)];

    // Each warp loops over pairs of atoms.

    for (int pair = warp; pair < *neighborCount; pair += numWarps) {
        int atom1 = neighbors[pair].x;
        int atom2 = neighbors[pair].y;
        float3 delta = neighborDeltas[pair];
        float r = sqrtf(delta.x*delta.x + delta.y*delta.y + delta.z*delta.z);
        float rInv = 1/r;

        // Compute the Gaussian basis functions and store them in temp1.

        int first = numGaussians, last = 0;
        for (int i = indexInWarp; i < numGaussians; i += 32) {
            float gaussianPos = i*cutoff/(numGaussians-1);
            float x = (r-gaussianPos)/gaussianWidth;
            float gaussian = expf(-0.5f*x*x);
            temp1[i] = gaussian;
            dtemp1[i] = -x*gaussian/gaussianWidth;
            if (temp1[i] > 1e-15) {
                if (first == numGaussians)
                    first = i;
                last = i;
            }
        }
        first = warpReduceMin(first);
        last = warpReduceMax(last);

        // Apply the first dense layer, storing the result in temp2.

        for (int i = indexInWarp; i < width; i += 32) {
            float sum = b1[i], dSumdR = 0;
            for (int j = first; j <= last; j++) {
                float w = w1[i+j*width];
                sum += temp1[j]*w;
                dSumdR += dtemp1[j]*w;
            }
            if (activation == 0) {
                float expSum = expf(sum);
                temp2[i] = logf(0.5f*expSum + 0.5f);
                dtemp2[i] = dSumdR*expSum/(expSum + 1);
            }
            else {
                float th = tanhf(sum);
                temp2[i] = th;
                dtemp2[i] = dSumdR*(1-th*th);
            }
        }
        __syncwarp();

        // Apply the second dense layer, storing the result in temp1.

        float cutoffScale = cutoffFunction(r, cutoff);
        float dCutoffdR = cutoffDeriv(r, cutoff);
        for (int i = indexInWarp; i < width; i += 32) {
            float sum = b2[i], dSumdR = 0;
            for (int j = 0; j < width; j++) {
                float w = w2[i+j*width];
                sum += temp2[j]*w;
                dSumdR += dtemp2[j]*w;
            }
            temp1[i] = cutoffScale*sum;
            dtemp1[i] = dCutoffdR*sum + cutoffScale*dSumdR;
        }
        __syncwarp();

        // Add it to the output.

        for (int i = indexInWarp; i < width; i += 32) {
            int index1 = atom1*width+i;
            int index2 = atom2*width+i;
            atomicAdd(&inputDeriv[index1], temp1[i]*outputDeriv[index2]);
            atomicAdd(&inputDeriv[index2], temp1[i]*outputDeriv[index1]);
            float scale = rInv*dtemp1[i]*(input[index2]*outputDeriv[index1] + input[index1]*outputDeriv[index2]);
            for (int offset = 16; offset > 0; offset /= 2)
                scale += __shfl_down_sync(0xFFFFFFFF, scale, offset);
            if (indexInWarp == 0) {
                float dVdX[3] = {scale*delta.x, scale*delta.y, scale*delta.z};
                for (int j = 0; j < 3; j++) {
                    atomicAdd(&positionDeriv[atom1*3+j], -dVdX[j]);
                    atomicAdd(&positionDeriv[atom2*3+j], dVdX[j]);
                }
            }
        }
    }
}


void CudaCFConv::backprop(const CFConvNeighbors& neighbors, const float* positions, const float* periodicBoxVectors,
                        const float* input, const float* outputDeriv, float* inputDeriv, float* positionDeriv) {
    // Get device pointers to all the data we need, copying it if necessary.

    const float* deviceInput = ensureOnDevice(input, this->input, getNumAtoms()*getWidth()*sizeof(float));
    const float* deviceOutputDeriv = ensureOnDevice(outputDeriv, this->output, getNumAtoms()*getWidth()*sizeof(float));
    float* deviceInputDeriv = ensureOnDevice(inputDeriv, this->inputDeriv, getNumAtoms()*getWidth()*sizeof(float));
    float* devicePositionDeriv = ensureOnDevice(positionDeriv, this->positionDeriv, getNumAtoms()*sizeof(float3));

    // Clear the output arrays.

    CHECK_RESULT(hipMemsetAsync(deviceInputDeriv, 0, getNumAtoms()*getWidth()*sizeof(float)));
    CHECK_RESULT(hipMemsetAsync(devicePositionDeriv, 0, getNumAtoms()*sizeof(float3)));

    // Invoke the kernel.

    const int blockSize = 512;
    const CudaCFConvNeighbors& cudaNeighbors = dynamic_cast<const CudaCFConvNeighbors&>(neighbors);
    const int numBlocks = numMultiprocessors*2;
    const int tempSize = max(getNumGaussians(), getWidth())*(blockSize/32);
    if (getPeriodic()) {
        if (neighbors.getTriclinic())
            backpropCFConv<true, true><<<numBlocks, blockSize, 4*tempSize*sizeof(float)>>>(getNumAtoms(), getNumGaussians(),
                getWidth(), getCutoff(), getGaussianWidth(), getActivation(), cudaNeighbors.getNeighbors(),
                cudaNeighbors.getNeighborCount(), cudaNeighbors.getNeighborDeltas(), deviceInput, deviceOutputDeriv,
                deviceInputDeriv, devicePositionDeriv, w1, b1, w2, b2);
        else
            backpropCFConv<true, false><<<numBlocks, blockSize, 4*tempSize*sizeof(float)>>>(getNumAtoms(), getNumGaussians(),
                getWidth(), getCutoff(), getGaussianWidth(), getActivation(), cudaNeighbors.getNeighbors(),
                cudaNeighbors.getNeighborCount(), cudaNeighbors.getNeighborDeltas(), deviceInput, deviceOutputDeriv,
                deviceInputDeriv, devicePositionDeriv, w1, b1, w2, b2);
    }
    else
        backpropCFConv<false, false><<<numBlocks, blockSize, 4*tempSize*sizeof(float)>>>(getNumAtoms(), getNumGaussians(),
            getWidth(), getCutoff(), getGaussianWidth(), getActivation(), cudaNeighbors.getNeighbors(),
            cudaNeighbors.getNeighborCount(), cudaNeighbors.getNeighborDeltas(), deviceInput, deviceOutputDeriv,
            deviceInputDeriv, devicePositionDeriv, w1, b1, w2, b2);

    // If necessary, copy the output.

    if (deviceInputDeriv == this->inputDeriv)
        CHECK_RESULT(hipMemcpy(inputDeriv, deviceInputDeriv, getNumAtoms()*getWidth()*sizeof(float), hipMemcpyDefault));
    if (devicePositionDeriv == this->positionDeriv)
        CHECK_RESULT(hipMemcpy(positionDeriv, devicePositionDeriv, getNumAtoms()*sizeof(float3), hipMemcpyDefault));
}
