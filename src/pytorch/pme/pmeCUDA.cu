#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <array>
#include <cmath>
#include <vector>

#include "common/accessor.cuh"

using namespace std;
using namespace torch::autograd;
using torch::Tensor;
using torch::TensorOptions;
using torch::Scalar;

#define CHECK_RESULT(result) \
    if (result != hipSuccess) { \
        throw runtime_error(string("Encountered error ")+hipGetErrorName(result)+" at "+__FILE__+":"+to_string(__LINE__));\
    }

__device__ void invertBoxVectors(const Accessor<float, 2>& box, float recipBoxVectors[3][3]) {
    float determinant = box[0][0]*box[1][1]*box[2][2];
    float scale = 1.0/determinant;
    recipBoxVectors[0][0] = box[1][1]*box[2][2]*scale;
    recipBoxVectors[0][1] = 0;
    recipBoxVectors[0][2] = 0;
    recipBoxVectors[1][0] = -box[1][0]*box[2][2]*scale;
    recipBoxVectors[1][1] = box[0][0]*box[2][2]*scale;
    recipBoxVectors[1][2] = 0;
    recipBoxVectors[2][0] = box[1][0]*box[2][1]-box[1][1]*box[2][0]*scale;
    recipBoxVectors[2][1] = -box[0][0]*box[2][1]*scale;
    recipBoxVectors[2][2] = box[0][0]*box[1][1]*scale;
}


__device__ void computeSpline(int atom, const Accessor<float, 2> pos, const Accessor<float, 2> box,
                          const float recipBoxVectors[3][3], const int gridSize[3], int gridIndex[3], float data[][3],
                          float ddata[][3], int pmeOrder) {
    // Find the position relative to the nearest grid point.

    float posInBox[3], t[3], dr[3];
    int ti[3];
    for (int i = 0; i < 3; i++)
         posInBox[i] = pos[atom][i]-box[i][i]*floor(pos[atom][i]*recipBoxVectors[i][i]);
    for (int i = 0; i < 3; i++) {
        t[i] = posInBox[0]*recipBoxVectors[0][i] + posInBox[1]*recipBoxVectors[1][i] + posInBox[2]*recipBoxVectors[2][i];
        t[i] = (t[i]-floor(t[i]))*gridSize[i];
        ti[i] = (int) t[i];
        dr[i] = t[i]-ti[i];
        gridIndex[i] = ti[i]%gridSize[i];
    }

    // Compute the B-spline coefficients.

    float scale = 1.0f/(pmeOrder-1);
    for (int i = 0; i < 3; i++) {
        data[pmeOrder-1][i] = 0;
        data[1][i] = dr[i];
        data[0][i] = 1-dr[i];
        for (int j = 3; j < pmeOrder; j++) {
            float div = 1.0f/(j-1);
            data[j-1][i] = div*dr[i]*data[j-2][i];
            for (int k = 1; k < j-1; k++)
                data[j-k-1][i] = div*((dr[i]+k)*data[j-k-2][i]+(j-k-dr[i])*data[j-k-1][i]);
            data[0][i] = div*(1-dr[i])*data[0][i];
        }
        if (ddata != NULL) {
            ddata[0][i] = -data[0][i];
            for (int j = 1; j < pmeOrder; j++)
                ddata[j][i] = data[j-1][i]-data[j][i];
        }
        data[pmeOrder-1][i] = scale*dr[i]*data[pmeOrder-2][i];
        for (int j = 1; j < pmeOrder-1; j++)
            data[pmeOrder-j-1][i] = scale*((dr[i]+j)*data[pmeOrder-j-2][i]+(pmeOrder-j-dr[i])*data[pmeOrder-j-1][i]);
        data[0][i] = scale*(1-dr[i])*data[0][i];
    }
}

template <int PME_ORDER>
__global__ void spreadCharge(const Accessor<float, 2> pos, const Accessor<float, 1> charge, const Accessor<float, 2> box,
                             Accessor<float, 3> grid, int gridx, int gridy, int gridz, float sqrtCoulomb) {
    float recipBoxVectors[3][3];
    invertBoxVectors(box, recipBoxVectors);
    float data[PME_ORDER][3];
    int numAtoms = pos.size(0);
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < numAtoms; atom += blockDim.x*gridDim.x) {
        int gridIndex[3];
        int gridSize[3] = {gridx, gridy, gridz};
        computeSpline(atom, pos, box,recipBoxVectors, gridSize, gridIndex, data, NULL, PME_ORDER);

        // Spread the charge from this atom onto each grid point.

        for (int ix = 0; ix < PME_ORDER; ix++) {
            int xindex = gridIndex[0]+ix;
            xindex -= (xindex >= gridx ? gridx : 0);
            float dx = charge[atom]*sqrtCoulomb*data[ix][0];
            for (int iy = 0; iy < PME_ORDER; iy++) {
                int yindex = gridIndex[1]+iy;
                yindex -= (yindex >= gridy ? gridy : 0);
                float dxdy = dx*data[iy][1];
                for (int iz = 0; iz < PME_ORDER; iz++) {
                    int zindex = gridIndex[2]+iz;
                    zindex -= (zindex >= gridz ? gridz : 0);
                    atomicAdd(&grid[xindex][yindex][zindex], dxdy*data[iz][2]);
                }
            }
        }
    }
}

__global__ void reciprocalConvolution(const Accessor<float, 2> box, Accessor<c10::complex<float>, 3> grid, int gridx, int gridy, int gridz,
                                      const Accessor<float, 1> xmoduli, const Accessor<float, 1> ymoduli, const Accessor<float, 1> zmoduli,
                                      float recipExpFactor, Accessor<float, 1> energyBuffer) {
    float recipBoxVectors[3][3];
    invertBoxVectors(box, recipBoxVectors);
    const unsigned int gridSize = gridx*gridy*(gridz/2+1);
    const float recipScaleFactor = recipBoxVectors[0][0]*recipBoxVectors[1][1]*recipBoxVectors[2][2]/M_PI;
    float energy = 0;

    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < gridSize; index += blockDim.x*gridDim.x) {
        int kx = index/(gridy*(gridz/2+1));
        int remainder = index-kx*gridy*(gridz/2+1);
        int ky = remainder/(gridz/2+1);
        int kz = remainder-ky*(gridz/2+1);
        int mx = (kx < (gridx+1)/2) ? kx : (kx-gridx);
        int my = (ky < (gridy+1)/2) ? ky : (ky-gridy);
        int mz = (kz < (gridz+1)/2) ? kz : (kz-gridz);
        float mhx = mx*recipBoxVectors[0][0];
        float mhy = mx*recipBoxVectors[1][0]+my*recipBoxVectors[1][1];
        float mhz = mx*recipBoxVectors[2][0]+my*recipBoxVectors[2][1]+mz*recipBoxVectors[2][2];
        float bx = xmoduli[kx];
        float by = ymoduli[ky];
        float bz = zmoduli[kz];
        c10::complex<float>& g = grid[kx][ky][kz];
        float m2 = mhx*mhx+mhy*mhy+mhz*mhz;
        float denom = m2*bx*by*bz;
        float eterm = (index == 0 ? 0 : recipScaleFactor*exp(-recipExpFactor*m2)/denom);
        float scale = (kz > 0 && kz <= (gridz-1)/2 ? 2 : 1);
        energy += scale * eterm * (g.real()*g.real() + g.imag()*g.imag());
        g *= eterm;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] = energy;
}

template <int PME_ORDER>
__global__ void interpolateForce(const Accessor<float, 2> pos, const Accessor<float, 1> charge, const Accessor<float, 2> box,
                                 const Accessor<float, 3> grid, int gridx, int gridy, int gridz, float sqrtCoulomb,
                                 Accessor<float, 2> posDeriv, Accessor<float, 1> chargeDeriv) {
    float recipBoxVectors[3][3];
    invertBoxVectors(box, recipBoxVectors);
    float data[PME_ORDER][3];
    float ddata[PME_ORDER][3];
    int numAtoms = pos.size(0);
    
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < numAtoms; atom += blockDim.x*gridDim.x) {
        int gridIndex[3];
        int gridSize[3] = {gridx, gridy, gridz};
        computeSpline(atom, pos, box,recipBoxVectors, gridSize, gridIndex, data, ddata, PME_ORDER);

        // Compute the derivatives on this atom.

        float dpos[3] = {0, 0, 0};
        float dq = 0;
        for (int ix = 0; ix < PME_ORDER; ix++) {
            int xindex = gridIndex[0]+ix;
            xindex -= (xindex >= gridx ? gridx : 0);
            float dx = data[ix][0];
            float ddx = ddata[ix][0];
            for (int iy = 0; iy < PME_ORDER; iy++) {
                int yindex = gridIndex[1]+iy;
                yindex -= (yindex >= gridy ? gridy : 0);
                float dy = data[iy][1];
                float ddy = ddata[iy][1];
                for (int iz = 0; iz < PME_ORDER; iz++) {
                    int zindex = gridIndex[2]+iz;
                    zindex -= (zindex >= gridz ? gridz : 0);
                    float dz = data[iz][2];
                    float ddz = ddata[iz][2];
                    float g = grid[xindex][yindex][zindex];
                    dpos[0] += ddx*dy*dz*g;
                    dpos[1] += dx*ddy*dz*g;
                    dpos[2] += dx*dy*ddz*g;
                    dq += dx*dy*dz*g;
                }
            }
        }
        float scale = charge[atom]*sqrtCoulomb;
        posDeriv[atom][0] = scale*(dpos[0]*gridSize[0]*recipBoxVectors[0][0]);
        posDeriv[atom][1] = scale*(dpos[0]*gridSize[0]*recipBoxVectors[1][0] + dpos[1]*gridSize[1]*recipBoxVectors[1][1]);
        posDeriv[atom][2] = scale*(dpos[0]*gridSize[0]*recipBoxVectors[2][0] + dpos[1]*gridSize[1]*recipBoxVectors[2][1] + dpos[2]*gridSize[2]*recipBoxVectors[2][2]);
        chargeDeriv[atom] = dq*sqrtCoulomb;
    }
}

class PmeFunctionCuda : public Function<PmeFunctionCuda> {
public:
    static Tensor forward(AutogradContext *ctx,
                          const Tensor& positions,
                          const Tensor& charges,
                          const Tensor& box_vectors,
                          const Scalar& gridx,
                          const Scalar& gridy,
                          const Scalar& gridz,
                          const Scalar& order,
                          const Scalar& alpha,
                          const Scalar& coulomb,
                          const Tensor& xmoduli,
                          const Tensor& ymoduli,
                          const Tensor& zmoduli) {
        const auto stream = c10::cuda::getCurrentCUDAStream(positions.get_device());
        const c10::cuda::CUDAStreamGuard guard(stream);
        int numAtoms = positions.size(0);
        int pmeOrder = (int) order.toInt();
        int gridSize[3] = {(int) gridx.toInt(), (int) gridy.toInt(), (int) gridz.toInt()};
        float sqrtCoulomb = sqrt(coulomb.toDouble());

        // Set an upper limit on how many thread blocks we try to launch based on the size of the GPU.

        int device, numMultiprocessors;
        CHECK_RESULT(hipGetDevice(&device));
        CHECK_RESULT(hipDeviceGetAttribute(&numMultiprocessors, hipDeviceAttributeMultiprocessorCount, device));
        int maxBlocks = numMultiprocessors*4;

        // Spread the charge on the grid.

        TensorOptions options = torch::TensorOptions().device(positions.device());
        Tensor realGrid = torch::zeros({gridSize[0], gridSize[1], gridSize[2]}, options);
        int blockSize = 128;
        int numBlocks = max(1, min(maxBlocks, numAtoms/blockSize));
        TORCH_CHECK(pmeOrder == 4 || pmeOrder == 5, "Only pmeOrder 4 or 5 is supported with CUDA");
        if (pmeOrder == 4)
            spreadCharge<4><<<numBlocks, blockSize, 0, stream>>>(get_accessor<float, 2>(positions), get_accessor<float, 1>(charges),
                    get_accessor<float, 2>(box_vectors), get_accessor<float, 3>(realGrid), gridSize[0], gridSize[1], gridSize[2], sqrtCoulomb);
        else
            spreadCharge<5><<<numBlocks, blockSize, 0, stream>>>(get_accessor<float, 2>(positions), get_accessor<float, 1>(charges),
                    get_accessor<float, 2>(box_vectors), get_accessor<float, 3>(realGrid), gridSize[0], gridSize[1], gridSize[2], sqrtCoulomb);

        // Take the Fourier transform.

        Tensor recipGrid = torch::fft::rfftn(realGrid);

        // Perform the convolution and calculate the energy.

        Tensor energy = torch::zeros(numBlocks*blockSize, options);
        reciprocalConvolution<<<numBlocks, blockSize, 0, stream>>>(get_accessor<float, 2>(box_vectors), get_accessor<c10::complex<float>, 3>(recipGrid),
                gridSize[0], gridSize[1], gridSize[2], get_accessor<float, 1>(xmoduli), get_accessor<float, 1>(ymoduli), get_accessor<float, 1>(zmoduli),
                M_PI*M_PI/(alpha.toDouble()*alpha.toDouble()), get_accessor<float, 1>(energy));

        // Store data for later use.

        ctx->save_for_backward({positions, charges, box_vectors, xmoduli, ymoduli, zmoduli, recipGrid});
        ctx->saved_data["gridx"] = gridx;
        ctx->saved_data["gridy"] = gridy;
        ctx->saved_data["gridz"] = gridz;
        ctx->saved_data["order"] = order;
        ctx->saved_data["alpha"] = alpha;
        ctx->saved_data["coulomb"] = coulomb;
        return {0.5*torch::sum(energy)};
    }

    static tensor_list backward(AutogradContext *ctx, tensor_list grad_outputs) {
        auto saved = ctx->get_saved_variables();
        Tensor positions = saved[0];
        Tensor charges = saved[1];
        Tensor box_vectors = saved[2];
        Tensor xmoduli = saved[3];
        Tensor ymoduli = saved[4];
        Tensor zmoduli = saved[5];
        Tensor recipGrid = saved[6];
        int gridSize[3] = {(int) ctx->saved_data["gridx"].toInt(), (int) ctx->saved_data["gridy"].toInt(), (int) ctx->saved_data["gridz"].toInt()};
        int pmeOrder = (int) ctx->saved_data["order"].toInt();
        float alpha = (float) ctx->saved_data["alpha"].toDouble();
        float sqrtCoulomb = sqrt(ctx->saved_data["coulomb"].toDouble());
        const auto stream = c10::cuda::getCurrentCUDAStream(positions.get_device());
        const c10::cuda::CUDAStreamGuard guard(stream);
        int numAtoms = positions.size(0);

        // Set an upper limit on how many thread blocks we try to launch based on the size of the GPU.

        int device, numMultiprocessors;
        CHECK_RESULT(hipGetDevice(&device));
        CHECK_RESULT(hipDeviceGetAttribute(&numMultiprocessors, hipDeviceAttributeMultiprocessorCount, device));
        int maxBlocks = numMultiprocessors*4;

        // Take the inverse Fourier transform.

        Tensor realGrid = torch::fft::irfftn(recipGrid)*(gridSize[0]*gridSize[1]*gridSize[2]);

        // Compute the derivatives.

        TensorOptions options = torch::TensorOptions().device(positions.device());
        Tensor posDeriv = torch::empty({numAtoms, 3}, options);
        Tensor chargeDeriv = torch::empty({numAtoms}, options);
        int blockSize = 128;
        int numBlocks = max(1, min(maxBlocks, numAtoms/blockSize));
        TORCH_CHECK(pmeOrder == 4 || pmeOrder == 5, "Only pmeOrder 4 or 5 is supported with CUDA");
        if (pmeOrder == 4)
            interpolateForce<4><<<numBlocks, blockSize, 0, stream>>>(get_accessor<float, 2>(positions), get_accessor<float, 1>(charges),
                        get_accessor<float, 2>(box_vectors), get_accessor<float, 3>(realGrid), gridSize[0], gridSize[1], gridSize[2], sqrtCoulomb,
                        get_accessor<float, 2>(posDeriv), get_accessor<float, 1>(chargeDeriv));
        else
            interpolateForce<5><<<numBlocks, blockSize, 0, stream>>>(get_accessor<float, 2>(positions), get_accessor<float, 1>(charges),
                        get_accessor<float, 2>(box_vectors), get_accessor<float, 3>(realGrid), gridSize[0], gridSize[1], gridSize[2], sqrtCoulomb,
                        get_accessor<float, 2>(posDeriv), get_accessor<float, 1>(chargeDeriv));
        posDeriv *= grad_outputs[0];
        chargeDeriv *= grad_outputs[0];
        torch::Tensor ignore;
        return {posDeriv, chargeDeriv, ignore, ignore, ignore, ignore, ignore, ignore, ignore, ignore, ignore, ignore};
    }
};

Tensor pme_reciprocal_cuda(const Tensor& positions,
                           const Tensor& charges,
                           const Tensor& box_vectors,
                           const Scalar& gridx,
                           const Scalar& gridy,
                           const Scalar& gridz,
                           const Scalar& order,
                           const Scalar& alpha,
                           const Scalar& coulomb,
                           const Tensor& xmoduli,
                           const Tensor& ymoduli,
                           const Tensor& zmoduli) {
    return PmeFunctionCuda::apply(positions, charges, box_vectors, gridx, gridy, gridz, order, alpha, coulomb, xmoduli, ymoduli, zmoduli);
}

TORCH_LIBRARY_IMPL(pme, AutogradCUDA, m) {
    m.impl("pme_reciprocal", pme_reciprocal_cuda);
}
