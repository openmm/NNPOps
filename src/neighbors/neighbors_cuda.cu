#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>
#include <algorithm>
#include <tuple>

using c10::cuda::CUDAStreamGuard;
using c10::cuda::getCurrentCUDAStream;
using std::make_tuple;
using std::max;
using torch::autograd::AutogradContext;
using torch::autograd::Function;
using torch::autograd::tensor_list;
using torch::empty;
using torch::full;
using torch::kInt32;
using torch::PackedTensorAccessor32;
using torch::RestrictPtrTraits;
using torch::Scalar;
using torch::Tensor;
using torch::TensorOptions;
using torch::zeros;

template <typename scalar_t, int num_dims>
    using Accessor = PackedTensorAccessor32<scalar_t, num_dims, RestrictPtrTraits>;

template <typename scalar_t, int num_dims> 
inline Accessor<scalar_t, num_dims> get_accessor(const Tensor& tensor) {
    return tensor.packed_accessor32<scalar_t, num_dims, RestrictPtrTraits>();
};

template <typename scalar_t> __device__ __forceinline__ scalar_t sqrt_(scalar_t x) {};
template<> __device__ __forceinline__ float sqrt_(float x) { return ::sqrtf(x); };
template<> __device__ __forceinline__ double sqrt_(double x) { return ::sqrt(x); };

template <typename scalar_t> __global__ void forward_kernel(
    const int32_t num_all_pairs,
    const Accessor<scalar_t, 2> positions,
    const scalar_t cutoff2,
    const bool store_all_pairs,
    Accessor<int32_t, 1> i_curr_pair,
    Accessor<int32_t, 2> neighbors,
    Accessor<scalar_t, 2> deltas,
    Accessor<scalar_t, 1> distances
) {
    const int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_all_pairs) return;

    int32_t row = floor((sqrtf(8 * index + 1) + 1) / 2);
    if (row * (row - 1) > 2 * index) row--;
    const int32_t column = index - row * (row - 1) / 2;

    const scalar_t delta_x = positions[row][0] - positions[column][0];
    const scalar_t delta_y = positions[row][1] - positions[column][1];
    const scalar_t delta_z = positions[row][2] - positions[column][2];
    const scalar_t distance2 = delta_x * delta_x + delta_y * delta_y + delta_z * delta_z;

    if (distance2 > cutoff2) return;

    const int32_t i_pair = store_all_pairs ? index : atomicAdd(&i_curr_pair[0], 1);

    neighbors[0][i_pair] = row;
    neighbors[1][i_pair] = column;
    deltas[i_pair][0] = delta_x;
    deltas[i_pair][1] = delta_y;
    deltas[i_pair][2] = delta_z;
    distances[i_pair] = sqrt_(distance2);
}

template <typename scalar_t> __global__ void backward_kernel(
    const Accessor<int32_t, 2> neighbors,
    const Accessor<scalar_t, 2> deltas,
    const Accessor<scalar_t, 1> distances,
    const Accessor<scalar_t, 1> grad_distances,
    Accessor<scalar_t, 2> grad_positions
) {
    const int32_t i_pair = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t num_pairs = neighbors.size(1);
    if (i_pair >= num_pairs) return;

    const int32_t i_dir = blockIdx.y;
    const int32_t i_atom = neighbors[i_dir][i_pair];
    if (i_atom < 0) return;

    const int32_t i_comp = blockIdx.z;
    const scalar_t grad = deltas[i_pair][i_comp] / distances[i_pair] * grad_distances[i_pair];
    atomicAdd(&grad_positions[i_atom][i_comp], (i_dir ? -1 : 1) * grad);
}

class Autograd : public Function<Autograd> {
public:
    static tensor_list forward(AutogradContext* ctx,
                               const Tensor& positions,
                               const Scalar& cutoff,
                               const Scalar& max_num_neighbors) {

        TORCH_CHECK(positions.dim() == 2, "Expected \"positions\" to have two dimensions");
        TORCH_CHECK(positions.size(0) > 0, "Expected the 1nd dimension size of \"positions\" to be more than 0");
        TORCH_CHECK(positions.size(1) == 3, "Expected the 2nd dimension size of \"positions\" to be 3");
        TORCH_CHECK(positions.is_contiguous(), "Expected \"positions\" to be contiguous");

        const int max_num_neighbors_ = max_num_neighbors.to<int>();
        TORCH_CHECK(max_num_neighbors_ > 0, "Expected \"max_num_neighbors\" to be positive");

        // Decide the algorithm
        const int num_atoms = positions.size(0);
        const int num_all_pairs = num_atoms * (num_atoms - 1) / 2;
        const int num_exp_pairs = num_atoms * max_num_neighbors_;
        const bool store_all_pairs = num_all_pairs <= num_exp_pairs;
        const int num_pairs = store_all_pairs ? num_all_pairs : num_exp_pairs;

        const int num_threads = 128;
        const int num_blocks = max((num_all_pairs + num_threads - 1) / num_threads, 1);
        const auto stream = getCurrentCUDAStream(positions.get_device());

        const TensorOptions options = positions.options();
        const Tensor i_curr_pair = store_all_pairs ? empty(1, options.dtype(kInt32)) :
                                                     zeros(1, options.dtype(kInt32));
        const Tensor neighbors = full({2, num_pairs}, -1, options.dtype(kInt32));
        const Tensor deltas = empty({num_pairs, 3}, options);
        const Tensor distances = full(num_pairs, 0, options);

        AT_DISPATCH_FLOATING_TYPES(positions.scalar_type(), "get_neighbor_pairs_forward", [&]() {
            const CUDAStreamGuard guard(stream);
            const scalar_t cutoff_ = cutoff.to<scalar_t>();
            TORCH_CHECK(cutoff_ > 0, "Expected \"cutoff\" to be positive");
            forward_kernel<<<num_blocks, num_threads, 0, stream>>>(
                num_all_pairs,
                get_accessor<scalar_t, 2>(positions),
                cutoff_ * cutoff_,
                store_all_pairs,
                get_accessor<int32_t, 1>(i_curr_pair),
                get_accessor<int32_t, 2>(neighbors),
                get_accessor<scalar_t, 2>(deltas),
                get_accessor<scalar_t, 1>(distances));
        });

        ctx->save_for_backward({neighbors, deltas, distances});
        ctx->saved_data["num_atoms"] = num_atoms;

        return {neighbors, distances};
    }

    static tensor_list backward(AutogradContext* ctx, tensor_list grad_inputs) {

        const Tensor grad_distances = grad_inputs[1];
        const int num_atoms = ctx->saved_data["num_atoms"].toInt();
        const int num_pairs = grad_distances.size(0);
        const int num_threads = 128;
        const int num_blocks_x = max((num_pairs + num_threads - 1) / num_threads, 1);
        const dim3 blocks(num_blocks_x, 2, 3);
        const auto stream = getCurrentCUDAStream(grad_distances.get_device());

        const tensor_list data = ctx->get_saved_variables();
        const Tensor neighbors = data[0];
        const Tensor deltas = data[1];
        const Tensor distances = data[2];
        const Tensor grad_positions = zeros({num_atoms, 3}, grad_distances.options());

        AT_DISPATCH_FLOATING_TYPES(grad_distances.scalar_type(), "get_neighbor_pairs_backward", [&]() {
            const CUDAStreamGuard guard(stream);
            backward_kernel<<<blocks, num_threads, 0, stream>>>(
                get_accessor<int32_t, 2>(neighbors),
                get_accessor<scalar_t, 2>(deltas),
                get_accessor<scalar_t, 1>(distances),
                get_accessor<scalar_t, 1>(grad_distances),
                get_accessor<scalar_t, 2>(grad_positions));
        });

        return {grad_positions, Tensor(), Tensor()};
      }
};

TORCH_LIBRARY_IMPL(neighbors, AutogradCUDA, m) {
    m.impl("get_neighbor_pairs",
        [](const Tensor& positions, const Scalar& cutoff, const Scalar& max_num_neighbors){
            const tensor_list results = Autograd::apply(positions, cutoff, max_num_neighbors);
            return make_tuple(results[0], results[1]);
    });
}